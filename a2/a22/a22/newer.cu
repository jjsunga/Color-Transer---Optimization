#include "hip/hip_runtime.h"

/***********************************************************
* Color Transfer Implementation
************************************************************
* This code is an implementation of the paper [Reinhard2001].
* The program transfers the color of one image (in this code
* reference image) to another image (in this code target image).
*
* usage: > ColorTransfer.exe [target image] [reference image]
*
* This code is this programmed by 'tatsy'. You can use this
* code for any purpose :-)
************************************************************/

#pragma warning( disable : 4996)
#include <iostream>
#include <iomanip>
#include <cstdio>
#include <cstdlib>
using namespace std;

#include <opencv2\opencv.hpp>

//CUDA
#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <>
#include "Color3d.h"

//A2 Added functions
#define BLOCK_SIZE 256

//KERNELS BEGIN - unoptimized version
//                            vec         mat         vec
__global__ void matvec_kernel(float* d_A, float* d_B, float* d_C, const int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	float sum = 0;
	if (tid < n)
	{
		for (int i = 0; i < n; ++i)
		{
			sum += d_A[i] * d_B[(i * n) + tid];
		}

		d_C[tid] = sum;
	}
}
//KERNELS END


void vecTransfer(float* h, Color3d* v)
{
	for (int j = 0; j < 3; ++j)
		h[j] = v->v[j]; //whats the point of making a class if you're just going to make everything public
}

//m.ptr<float>(0); // is a pointer to the beginning of the row in the matrix might be useful
void matTransfer(float* h, const cv::Mat& m)
{
	for (int i = 0; i < 3; ++i)
		for (int j = 0; i < 3; ++j)
			h[i * j + 3] = m.at<float>(i, j);
}

int math(int a, int b) {

	return a + b;
}

//KERNEL Helper function does setup and launch
void matvec_L(Color3d* vec, cv::Mat* mat, float* h_C)
{
	float *h_A, *h_B;
	float *d_A, *d_B, *d_C;

	int N = 3;
	h_A = (float*)malloc(sizeof(float) * N);
	h_B = new float[mat->total()];
	//h_A = new float(N);
	//float* mPtr = new float[mat->total()];

	hipMalloc((void**)&d_A, sizeof(float) * N);
	hipMalloc((void**)&d_B, sizeof(float) * N * N);
	hipMalloc((void**)&d_C, sizeof(float) * N);

	//copy vec and matrix to host pointers
	vecTransfer(h_A, vec);
	memcpy(h_B, mat->data, mat->total());



	hipMemcpy(d_A, h_A, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, sizeof(float) * N * N, hipMemcpyHostToDevice);

	matvec_kernel << <N / BLOCK_SIZE + 1, BLOCK_SIZE >> >(d_A, d_B, d_C, N);
	//printf("error code: %s\n",hipGetErrorString(hipGetLastError()));

	hipMemcpy(h_C, d_C, sizeof(float) * N, hipMemcpyDeviceToHost);

	free(h_A);
	free(h_B);
	//delete[] h_A;
	//delete[] h_B;

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);


}
// End of A2 added functions

// NOTE(marko) : most instances have been replaced but leave just incase
// Multiplication of matrix and vector
Color3d operator *(const cv::Mat& M, Color3d& v) {
	Color3d u = Color3d();
	for (int i = 0; i < 3; i++) {
		u(i) = 0.0;
		for (int j = 0; j < 3; j++) {
			u(i) += M.at<double>(i, j) * v(j);
		}
	}
	return u;
}

// Transformation from RGB to LMS
const double RGB2LMS[3][3] = {
	{ 0.3811, 0.5783, 0.0402 },
	{ 0.1967, 0.7244, 0.0782 },
	{ 0.0241, 0.1288, 0.8444 }
};

// Transformation from LMS to RGB
const double LMS2RGB[3][3] = {
	{ 4.4679, -3.5873,  0.1193 },
	{ -1.2186,  2.3809, -0.1624 },
	{ 0.0497, -0.2439,  1.2045 }
};

// First transformation from LMS to lab
const double LMS2lab1[3][3] = {
	{ 1.0 / sqrt(3.0), 0.0, 0.0 },
	{ 0.0, 1.0 / sqrt(6.0), 0.0 },
	{ 0.0, 0.0, 1.0 / sqrt(2.0) }
};

// Second transformation from LMS to lab
const double LMS2lab2[3][3] = {
	{ 1.0,  1.0,  1.0 },
	{ 1.0,  1.0, -2.0 },
	{ 1.0, -1.0,  0.0 }
};

const double eps = 1.0e-4;

int main(int argc, char** argv) {
	// Check number of arguments
	if (argc <= 2) {
		cout << "usage: > ColorTransfer.exe [target image] [reference image]" << endl;
		return -1;
	}

	//Easy transfer back
	//--------A2-----------
	float *h_C;
	int N = 3;
	h_C = (float*)malloc(sizeof(float) * N);
	//--------A2-----------

	// Load target image
	cv::Mat target = cv::imread(argv[1], CV_LOAD_IMAGE_COLOR);
	if (target.empty()) {
		cout << "Failed to load file \"" << argv[1] << "\"" << endl;
		return -1;
	}
	cv::cvtColor(target, target, CV_BGR2RGB);
	target.convertTo(target, CV_64FC3, 1.0 / 255.0);

	// Load reference image
	cv::Mat refer = cv::imread(argv[2], CV_LOAD_IMAGE_COLOR);
	if (refer.empty()) {
		cout << "Failed to load file \"" << argv[2] << "\"" << endl;
		return -1;
	}
	cv::cvtColor(refer, refer, CV_BGR2RGB);
	refer.convertTo(refer, CV_64FC3, 1.0 / 255.0);

	// Construct transformation matrix
	const size_t bufsize = sizeof(double) * 3 * 3;
	cv::Mat mRGB2LMS = cv::Mat(3, 3, CV_64FC1);
	memcpy(mRGB2LMS.data, &RGB2LMS[0][0], bufsize);

	cv::Mat mLMS2RGB = cv::Mat(3, 3, CV_64FC1);
	memcpy(mLMS2RGB.data, &LMS2RGB[0][0], bufsize);

	cv::Mat mLMS2lab1 = cv::Mat(3, 3, CV_64FC1);
	memcpy(mLMS2lab1.data, &LMS2lab1[0][0], bufsize);

	cv::Mat mLMS2lab2 = cv::Mat(3, 3, CV_64FC1);
	memcpy(mLMS2lab2.data, &LMS2lab2[0][0], bufsize);

	cv::Mat mLMS2lab = mLMS2lab2 * mLMS2lab1;
	cv::Mat mlab2LMS = mLMS2lab.inv();

	// Transform images from RGB to lab and
	// compute average and standard deviation of each color channels
	Color3d v;
	Color3d mt = Color3d(0.0, 0.0, 0.0);
	Color3d st = Color3d(0.0, 0.0, 0.0);
	for (int y = 0; y < target.rows; y++) {
		for (int x = 0; x < target.cols; x++) {
			v = target.at<Color3d>(y, x);

			matvec_L(&v, &mRGB2LMS, h_C);//Perform kernel launch and store result in h_C
			memcpy(&v, h_C, N * sizeof(float));// Transfer result back to v

			for (int c = 0; c < 3; c++)
				v(c) = v(c) > eps ? log10(v(c)) : log10(eps);

			matvec_L(&v, &mLMS2lab, h_C);//Perform kernel launch and store result in h_C
			memcpy(&target.at<Color3d>(y, x), h_C, N * sizeof(float));// Transfer result back to v

			mt = mt + target.at<Color3d>(y, x);
			st = st + target.at<Color3d>(y, x) * target.at<Color3d>(y, x);//maybe do this one too
		}
	}

	Color3d mr = Color3d(0.0, 0.0, 0.0);
	Color3d sr = Color3d(0.0, 0.0, 0.0);
	for (int y = 0; y < refer.rows; y++) {
		for (int x = 0; x < refer.cols; x++) {
			v = refer.at<Color3d>(y, x);

			matvec_L(&v, &mRGB2LMS, h_C);//Perform kernel launch and store result in h_C
			memcpy(&v, h_C, N * sizeof(float));// Transfer result back to v

			for (int c = 0; c < 3; c++)
				v(c) = v(c) > eps ? log10(v(c)) : log10(eps);

			matvec_L(&v, &mLMS2lab, h_C);//Perform kernel launch and store result in h_C
			memcpy(&refer.at<Color3d>(y, x), h_C, N * sizeof(float));// Transfer result back to v

			mr = mr + refer.at<Color3d>(y, x);
			sr = sr + refer.at<Color3d>(y, x) * refer.at<Color3d>(y, x);//maybe do this one too
		}
	}

	int Nt = target.rows * target.cols;
	int Nr = refer.rows * refer.cols;
	mt = mt.divide(Nt);
	mr = mr.divide(Nr);
	st = st.divide(Nt) - mt * mt;
	sr = sr.divide(Nr) - mr * mr;
	for (int i = 0; i < 3; i++) {
		st(i) = sqrt(st(i));
		sr(i) = sqrt(sr(i));
	}

	// Transfer colors
	for (int y = 0; y < target.rows; y++) {
		for (int x = 0; x < target.cols; x++) {
			for (int c = 0; c < 3; c++) {
				double val = target.at<double>(y, x * 3 + c);
				target.at<double>(y, x * 3 + c) = (val - mt(c)) / st(c) * sr(c) + mr(c);
			}
		}
	}

	// allocate host memory
	h_C = new float[3]; //result

						// Transform back from lab to RGB
	for (int y = 0; y < target.rows; y++) {
		for (int x = 0; x < target.cols; x++) {
			v = target.at<Color3d>(y, x);

			//Not sure if all results are stored back into v so I'm leaving it as two seperate calls
			matvec_L(&v, &mlab2LMS, h_C);//Perform kernel launch and store result in h_C
			memcpy(&v, h_C, N * sizeof(float));// Transfer result back to v

			for (int c = 0; c < 3; c++)
				v(c) = v(c) > -5.0 ? pow(10.0, v(c)) : eps;

			matvec_L(&v, &mLMS2RGB, h_C);//Perform kernel launch and store result in h_C
			memcpy(&target.at<Color3d>(y, x), h_C, N * sizeof(float));// Transfer result back to v
		}
	}

	target.convertTo(target, CV_8UC3, 255.0);
	cv::cvtColor(target, target, CV_RGB2BGR);

	//A2
	//free(h_C);
	delete[] h_C;

	cv::namedWindow("target");
	cv::imshow("target", target);
	cv::imwrite("output.jpg", target);
	cv::waitKey(0);
	cv::destroyAllWindows();
}
