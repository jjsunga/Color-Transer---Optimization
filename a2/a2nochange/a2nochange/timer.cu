#include "hip/hip_runtime.h"
/***********************************************************
* Color Transfer Implementation
************************************************************
* This code is an implementation of the paper [Reinhard2001].
* The program transfers the color of one image (in this code
* reference image) to another image (in this code target image).
*
* usage: > ColorTransfer.exe [target image] [reference image]
*
* This code is this programmed by 'tatsy'. You can use this
* code for any purpose :-)
************************************************************/

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <chrono>
#include <hip/hip_runtime.h>
#include ""
using namespace std::chrono;

#include <opencv2\opencv.hpp>

#include "Color3d.h"

void reportTime(const char* msg, steady_clock::duration span) {
	auto ms = duration_cast<milliseconds>(span);
	std::cout << msg << " - took - " <<
		ms.count() << " millisecs" << std::endl;
}

// Multiplication of matrix and vector
Color3d operator *(const cv::Mat& M, Color3d& v) {
	Color3d u = Color3d();
	for (int i = 0; i<3; i++) {
		u(i) = 0.0;
		for (int j = 0; j<3; j++) {
			u(i) += M.at<double>(i, j) * v(j);
		}
	}
	return u;
}

// Transformation from RGB to LMS
const double RGB2LMS[3][3] = {
	{ 0.3811, 0.5783, 0.0402 },
	{ 0.1967, 0.7244, 0.0782 },
	{ 0.0241, 0.1288, 0.8444 }
};

// Transformation from LMS to RGB
const double LMS2RGB[3][3] = {
	{ 4.4679, -3.5873,  0.1193 },
	{ -1.2186,  2.3809, -0.1624 },
	{ 0.0497, -0.2439,  1.2045 }
};

// First transformation from LMS to lab
const double LMS2lab1[3][3] = {
	{ 1.0 / sqrt(3.0), 0.0, 0.0 },
	{ 0.0, 1.0 / sqrt(6.0), 0.0 },
	{ 0.0, 0.0, 1.0 / sqrt(2.0) }
};

// Second transformation from LMS to lab
const double LMS2lab2[3][3] = {
	{ 1.0,  1.0,  1.0 },
	{ 1.0,  1.0, -2.0 },
	{ 1.0, -1.0,  0.0 }
};

const double eps = 1.0e-4;

int main(int argc, char** argv) {
	// Check number of arguments
	if (argc <= 2) {
		std::cout << "usage: > ColorTransfer.exe [target image] [reference image]" << std::endl;
		return -1;
	}

	// Load target image
	cv::Mat target = cv::imread(argv[1], CV_LOAD_IMAGE_COLOR);
	if (target.empty()) {
		std::cout << "Failed to load file \"" << argv[1] << "\"" << std::endl;
		return -1;
	}
	cv::cvtColor(target, target, CV_BGR2RGB);
	target.convertTo(target, CV_64FC3, 1.0 / 255.0);

	// Load reference image
	cv::Mat refer = cv::imread(argv[2], CV_LOAD_IMAGE_COLOR);
	if (refer.empty()) {
		std::cout << "Failed to load file \"" << argv[2] << "\"" << std::endl;
		return -1;
	}
	steady_clock::time_point ts, te;
	ts = steady_clock::now();

	cv::cvtColor(refer, refer, CV_BGR2RGB);
	refer.convertTo(refer, CV_64FC3, 1.0 / 255.0);

	// Construct transformation matrix
	const size_t bufsize = sizeof(double) * 3 * 3;
	cv::Mat mRGB2LMS = cv::Mat(3, 3, CV_64FC1);
	memcpy(mRGB2LMS.data, &RGB2LMS[0][0], bufsize);

	cv::Mat mLMS2RGB = cv::Mat(3, 3, CV_64FC1);
	memcpy(mLMS2RGB.data, &LMS2RGB[0][0], bufsize);

	cv::Mat mLMS2lab1 = cv::Mat(3, 3, CV_64FC1);
	memcpy(mLMS2lab1.data, &LMS2lab1[0][0], bufsize);

	cv::Mat mLMS2lab2 = cv::Mat(3, 3, CV_64FC1);
	memcpy(mLMS2lab2.data, &LMS2lab2[0][0], bufsize);

	cv::Mat mLMS2lab = mLMS2lab2 * mLMS2lab1;
	cv::Mat mlab2LMS = mLMS2lab.inv();

	// Transform images from RGB to lab and
	// compute average and standard deviation of each color channels
	Color3d v;
	Color3d mt = Color3d(0.0, 0.0, 0.0);
	Color3d st = Color3d(0.0, 0.0, 0.0);
	for (int y = 0; y<target.rows; y++) {
		for (int x = 0; x<target.cols; x++) {
			v = target.at<Color3d>(y, x);
			v = mRGB2LMS * v;
			for (int c = 0; c<3; c++) v(c) = v(c) > eps ? log10(v(c)) : log10(eps);

			target.at<Color3d>(y, x) = mLMS2lab * v;
			mt = mt + target.at<Color3d>(y, x);
			st = st + target.at<Color3d>(y, x) * target.at<Color3d>(y, x);
		}
	}

	Color3d mr = Color3d(0.0, 0.0, 0.0);
	Color3d sr = Color3d(0.0, 0.0, 0.0);
	for (int y = 0; y<refer.rows; y++) {
		for (int x = 0; x<refer.cols; x++) {
			v = refer.at<Color3d>(y, x);
			v = mRGB2LMS * v;
			for (int c = 0; c<3; c++) v(c) = v(c) > eps ? log10(v(c)) : log10(eps);

			refer.at<Color3d>(y, x) = mLMS2lab * v;
			mr = mr + refer.at<Color3d>(y, x);
			sr = sr + refer.at<Color3d>(y, x) * refer.at<Color3d>(y, x);
		}
	}

	int Nt = target.rows * target.cols;
	int Nr = refer.rows * refer.cols;
	mt = mt.divide(Nt);
	mr = mr.divide(Nr);
	st = st.divide(Nt) - mt * mt;
	sr = sr.divide(Nr) - mr * mr;
	for (int i = 0; i<3; i++) {
		st(i) = sqrt(st(i));
		sr(i) = sqrt(sr(i));
	}

	// Transfer colors
	for (int y = 0; y<target.rows; y++) {
		for (int x = 0; x<target.cols; x++) {
			for (int c = 0; c<3; c++) {
				double val = target.at<double>(y, x * 3 + c);
				target.at<double>(y, x * 3 + c) = (val - mt(c)) / st(c) * sr(c) + mr(c);
			}
		}
	}

	// Transform back from lab to RGB
	for (int y = 0; y<target.rows; y++) {
		for (int x = 0; x<target.cols; x++) {
			v = target.at<Color3d>(y, x);
			v = mlab2LMS * v;
			for (int c = 0; c<3; c++) v(c) = v(c) > -5.0 ? pow(10.0, v(c)) : eps;

			target.at<Color3d>(y, x) = mLMS2RGB * v;
		}
	}
	target.convertTo(target, CV_8UC3, 255.0);
	cv::cvtColor(target, target, CV_RGB2BGR);

	te = steady_clock::now();
	reportTime("target conversion took", te - ts);
	cv::namedWindow("target");
	cv::imshow("target", target);
	cv::imwrite("output.jpg", target);
	cv::waitKey(0);
	cv::destroyAllWindows();
}
